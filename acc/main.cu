#include "hip/hip_runtime.h"
#include <math.h>
#include <time.h>

#include "check.h"
extern "C"
{
#include "io.h"
#include "seq.h"
}

#define N 10
#define BLOCK_SIZE 256

__global__ void accumulate(float *a, float *b, int n)
{
	int i;

	i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
		a[i] += b[i];
}

int main(void)
{
	struct
	{
		float *h, *d;
	} a, b;
	struct timespec t;

	a.h = geometric(M_PI, M_E, N);
	b.h = geometric(M_SQRT2, M_LN2, N);
#if N <= 10
	print(a.h, N);
	print(b.h, N);
#endif
	CHECK_CUDA(hipMalloc(&a.d, N * sizeof(float)));
	CHECK_CUDA(hipMalloc(&b.d, N * sizeof(float)));
	CHECK_CUDA(hipMemcpy(a.d, a.h, N * sizeof(float), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(b.d, b.h, N * sizeof(float), hipMemcpyHostToDevice));
	clock_gettime(CLOCK_MONOTONIC, &t);
	accumulate<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(a.d, b.d, N);
        CHECK_CUDA(hipDeviceSynchronize());
	time_(t);
	CHECK_CUDA(hipMemcpy(a.h, a.d, N * sizeof(float), hipMemcpyDeviceToHost));
#if N <= 10
	print(a.h, N);
#endif
	return 0;
}
