#include "hip/hip_runtime.h"
#include <math.h>
#include <time.h>

#include "check.h"
extern "C"
{
#include "io.h"
#include "seq.h"
}

#define N 10
#define BLOCK_SIZE 256

__global__ void accumulate(float *a, float *b, int n)
{
	// TODO: Implement the accumulation on the GPU.
}

int main(void)
{
	struct
	{
		float *h, *d;
	} a, b;
	struct timespec t;

	a.h = geometric(M_PI, M_E, N);
	b.h = geometric(M_SQRT2, M_LN2, N);
#if N <= 10
	print(a.h, N);
	print(b.h, N);
#endif
	// TODO: Allocate a.d and b.d on the GPU.
	// TODO: Copy a.h to a.d and b.h to b.d.
	clock_gettime(CLOCK_MONOTONIC, &t);
	// TODO: Launch the kernel.
	// TODO: Synchronize with the device.
	time_(t);
	// TODO: Copy the result back to the host.
#if N <= 10
	print(a.h, N);
#endif
	return 0;
}
