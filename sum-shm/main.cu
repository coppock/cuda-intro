#include "hip/hip_runtime.h"
#include <stdio.h>

#include "check.h"
extern "C"
{
#include "io.h"
#include "seq.h"
}

#define N 10
#define BLOCK_SIZE 512

__global__ void sum(float *a, int n)
{
	// TODO: Implement the summation on the GPU using shared memory.
}

int main(void)
{
	struct
	{
		float *h, *d;
	} a;
	struct timespec t;
	int k;

	a.h = geometric(M_PI, M_LN2, N);
	// TODO: Allocate a.d on the GPU.
	// TODO: Copy a.h to a.d.
	clock_gettime(CLOCK_MONOTONIC, &t);
	// TODO: Launch the kernel repeatedly until we have the final sum.
	// TODO: Synchronize with the device.
	time_(t);
	// TODO: Copy the result back to the host.
	print(a.h, 1);
	return 0;
}
