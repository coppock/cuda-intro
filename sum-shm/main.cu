#include "hip/hip_runtime.h"
#include <stdio.h>

#include "check.h"
extern "C"
{
#include "io.h"
#include "seq.h"
}

#define N 100 * MEGA
#define BLOCK_SIZE 512

__global__ void sum(float *a, int n)
{
	int i;
	__shared__ float s[BLOCK_SIZE];

	i = blockIdx.x * blockDim.x + threadIdx.x;
	s[threadIdx.x] = a[i];
	__syncthreads();
	for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
	{
		if (threadIdx.x < stride)
			s[threadIdx.x] += s[threadIdx.x + stride];
		__syncthreads();
	}
	if (threadIdx.x == 0)
		a[blockIdx.x] = s[0];
}

int main(void)
{
	struct
	{
		float *h, *d;
	} a;
	struct timespec t;
	int k;

	a.h = geometric(M_PI, M_LN2, N);
	CHECK_CUDA(hipMalloc(&a.d, N * sizeof(float)));
	CHECK_CUDA(hipMemcpy(a.d, a.h, N * sizeof(float), hipMemcpyHostToDevice));
	clock_gettime(CLOCK_MONOTONIC, &t);
	for (k = N; k > 1; k = (k - 1) / BLOCK_SIZE + 1)
	{
		if (k <= 10)
			print(a.h, k);
		sum<<<(k - 1) / BLOCK_SIZE + 1, k / 2 > BLOCK_SIZE ? BLOCK_SIZE : k / 2>>>(a.d, k);
	}
	time_(t);
	CHECK_CUDA(hipMemcpy(a.h, a.d, N * sizeof(float), hipMemcpyDeviceToHost));
	print(a.h, 1);
	return 0;
}
