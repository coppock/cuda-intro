#include "hip/hip_runtime.h"
#include <stdio.h>

#include "check.h"
extern "C"
{
#include "io.h"
#include "seq.h"
}

#define N 100 * MEGA
#define BLOCK_SIZE 256

__global__ void sum(float *a, int n)
{
	int i;

	i = blockIdx.x * blockDim.x + threadIdx.x;
	for (; n > gridDim.x; n = (n + 1) / 2)
	{
		if (i < (n + 1) / 2)
			a[i] += a[i + (n + 1) / 2];
		__syncthreads();
	}
}

int main(void)
{
	struct
	{
		float *h, *d;
	} a;
	struct timespec t;
	int k;

	a.h = geometric(M_PI, M_LN2, N);
	CHECK_CUDA(hipMalloc(&a.d, N * sizeof(float)));
	CHECK_CUDA(hipMemcpy(a.d, a.h, N * sizeof(float), hipMemcpyHostToDevice));
	clock_gettime(CLOCK_MONOTONIC, &t);
	for (k = N; k > 1; k = (k - 1) / BLOCK_SIZE / 2 + 1)
	{
		if (k <= 10)
			print(a.h, k);
		sum<<<(k / 2 - 1) / BLOCK_SIZE + 1, k / 2 > BLOCK_SIZE ? BLOCK_SIZE : k / 2>>>(a.d, k);
	}
	time_(t);
	CHECK_CUDA(hipMemcpy(a.h, a.d, N * sizeof(float), hipMemcpyDeviceToHost));
	print(a.h, 1);
	return 0;
}
